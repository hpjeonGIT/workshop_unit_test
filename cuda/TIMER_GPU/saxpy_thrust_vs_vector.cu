#include <iostream>
#include <iterator>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <algorithm>
#include <vector>
#include <chrono>
#include <hip/hip_runtime_api.h>
// Ref: http://www.training.prace-ri.eu/uploads/tx_pracetmo/CUDAGC.pdf
class Timer_CUDA {
private:
    hipEvent_t start, stop;
public:
    Timer_CUDA() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
    }
    float elapsed() {
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float milliseconds = 0.0f;
	hipEventElapsedTime(&milliseconds, start, stop);
	return milliseconds;
    }
    ~Timer_CUDA() = default;
};
#define TIMER_GPU(f, etime, n) {Timer_CUDA t;for (uint i=0;i<n;i++) f; etime = t.elapsed(); }

// 
class Timer {
private:    
    std::chrono::time_point<std::chrono::steady_clock> start, end;
public:    
    Timer() {
	start = std::chrono::steady_clock::now();
    }
    float elapsed() {
	end = std::chrono::steady_clock::now();
	return static_cast<float>  (std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());
    }
    ~ Timer() = default;
};
    
#define TIMER_CPU(fn, etime, n) { Timer t; for (uint i=0;i<n; i++) fn; etime= t.elapsed() ;}
    

void saxpy1(float A, thrust::device_vector<float>& X,
	   thrust::device_vector<float>& Y) {
    thrust::device_vector<float> temp(X.size());
    thrust::fill(temp.begin(), temp.end(), A);
    thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(),
		      thrust::multiplies<float>());
    thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(),
		      thrust::plus<float>());
}

struct saxpy_functor
{
    const float a;
    saxpy_functor(float _a) : a(_a) {}
    __host__ __device__ float operator() (const float& x, const float& y) const {
	return a*x + y;
    }
};

void saxpy2(float A, thrust::device_vector<float>& X,
	    thrust::device_vector<float>& Y) {
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(),
		      saxpy_functor(A));
}

void saxpy_cpu(float A, std::vector<float>& X, std::vector<float>& Y) {
    for (uint i=0; i<X.size(); i++) {
	Y[i] = A*X[i] + Y[i];
    }
    
}
int main(void)
{
    const int N=1000000;
    thrust::host_vector<float> tmp(N);
    thrust::generate(tmp.begin(), tmp.end(), rand);
    thrust::device_vector<float> X=tmp, Y=tmp;
    float etime1, etime2;

    // Warming up GPU card
    TIMER_GPU(saxpy1(10.f, X, Y), etime1, 100);
    // Actual tests
    TIMER_GPU(saxpy1(10.f, X, Y), etime1, 100);
    std::cout << etime1 << std::endl;
    TIMER_GPU(saxpy2(10.f, X, Y), etime2, 100);
    std::cout << etime2 << std::endl;

    //CPU
    std::vector<float> Xc(N);
    std::generate(Xc.begin(), Xc.end(), rand);
    std::vector<float> Yc = Xc;
    TIMER_CPU(saxpy_cpu(10.f, Xc, Yc), etime1, 100);
    std::cout << etime1 <<std::endl;
    
    return 0;
}
    
