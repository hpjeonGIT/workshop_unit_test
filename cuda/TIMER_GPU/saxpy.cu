#include <thrust/transform.h>
#include <thrust/functional.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
// Ref: http://www.training.prace-ri.eu/uploads/tx_pracetmo/CUDAGC.pdf
class Timer {
private:
    hipEvent_t start, stop;
public:
    Timer() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
    }
    float elapsed() {
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float milliseconds = 0.0f;
	hipEventElapsedTime(&milliseconds, start, stop);
	return milliseconds;
    }
    ~Timer() = default;
};
#define TIMER_GPU(f, etime, n) {Timer t;for (uint i=0;i<n;i++) f; etime = t.elapsed(); }
    
void saxpy1(float A, thrust::device_vector<float>& X,
	   thrust::device_vector<float>& Y) {
    thrust::device_vector<float> temp(X.size());
    thrust::fill(temp.begin(), temp.end(), A);
    thrust::transform(X.begin(), X.end(), temp.begin(), temp.begin(),
		      thrust::multiplies<float>());
    thrust::transform(temp.begin(), temp.end(), Y.begin(), Y.begin(),
		      thrust::plus<float>());
}

struct saxpy_functor
{
    const float a;
    saxpy_functor(float _a) : a(_a) {}
    __host__ __device__ float operator() (const float& x, const float& y) const {
	return a*x + y;
    }
};

void saxpy2(float A, thrust::device_vector<float>& X,
	    thrust::device_vector<float>& Y) {
    thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(),
		      saxpy_functor(A));
}

int main(void)
{
    const int N=1000000;
    thrust::host_vector<float> tmp(N);
    thrust::generate(tmp.begin(), tmp.end(), rand);
    thrust::device_vector<float> X=tmp, Y=tmp;
    float etime1, etime2;

    // Warming up GPU card
    TIMER_GPU(saxpy1(10.f, X, Y), etime1, 10);
    // Actual tests
    TIMER_GPU(saxpy1(10.f, X, Y), etime1, 10);
    std::cout << etime1 << std::endl;
    TIMER_GPU(saxpy2(10.f, X, Y), etime2, 10);
    std::cout << etime2 << std::endl;
 
    
    return 0;
}
    
