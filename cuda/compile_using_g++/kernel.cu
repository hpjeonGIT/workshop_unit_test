#include "hip/hip_runtime.h"
// kernel.cu
#include <kernel.h>

__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int cuda_vec_add(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, n*sizeof(float));
    hipMalloc(&d_b, n*sizeof(float));
    hipMalloc(&d_c, n*sizeof(float));

    hipMemcpy(d_a, h_a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n*sizeof(float), hipMemcpyHostToDevice);

    vec_add_kernel<< <(n-1)/256+1,256>> >(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
