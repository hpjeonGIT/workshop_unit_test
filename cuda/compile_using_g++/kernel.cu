#include "hip/hip_runtime.h"
// kernel.cu
#include <kernel.h>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

__global__ void vec_add_kernel(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int cuda_vec_add(float *h_a, float *h_b, float *h_c, int n) {
    float *d_a, *d_b, *d_c;

    checkCudaErrors(hipMalloc(&d_a, n*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_b, n*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_c, n*sizeof(float)));

    checkCudaErrors(hipMemcpy(d_a, h_a, n*sizeof(float),
    				    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, h_b, n*sizeof(float),
    				    hipMemcpyHostToDevice));

    vec_add_kernel<< <(n-1)/256+1,256>> >(d_a, d_b, d_c, n);

    checkCudaErrors(hipMemcpy(h_c, d_c, n*sizeof(float),
    				    hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));

    return 0;
}
